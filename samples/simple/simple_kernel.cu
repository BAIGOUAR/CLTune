#include "hip/hip_runtime.h"

// =================================================================================================
// This file is part of the CLTune project, which loosely follows the Google C++ styleguide and uses
// a tab-size of two spaces and a max-width of 100 characters per line.
//
// Author(s):
//   Cedric Nugteren <www.cedricnugteren.nl>
//
// This file contains an example CUDA kernel as part of the simple_kernel.cc example.
//
// =================================================================================================

extern "C" __global__ void vector_add(const int n, float *a, float *b, float *c) {
  const int i = blockIdx.x * GROUP_SIZE + threadIdx.x;
  if (i < n) {
    c[i] = a[i] + b[i];
  }
}

// =================================================================================================
